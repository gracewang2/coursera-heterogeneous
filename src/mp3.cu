#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

const int TILE_WIDTH = 16;

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows,
                                     int numAColumns, int numBRows,
                                     int numBColumns, int numCRows,
                                     int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float Cvalue = 0;

  // Loop over the A and B tiles required to compute the C element
  for (int c = 0; c < (numAColumns - 1)/TILE_WIDTH + 1; c++) {
    // Collaborative loading of A and B tiles into shared memory
    if (row < numARows && c * TILE_WIDTH + threadIdx.x < numAColumns) {
      ds_A[threadIdx.y][threadIdx.x] = A[row * numAColumns + c * TILE_WIDTH + threadIdx.x];
    }
    else ds_A[threadIdx.y][threadIdx.x] = 0;

    if (c * TILE_WIDTH + threadIdx.y < numBRows && col < numBColumns) {
      ds_B[threadIdx.y][threadIdx.x] = B[(c * TILE_WIDTH + threadIdx.y) * numBColumns + col];
    } 
    else ds_B[threadIdx.y][threadIdx.x] = 0;
    __syncthreads();

    for (int i = 0; i < TILE_WIDTH; i++) {
      Cvalue += ds_A[threadIdx.y][i] * ds_B[i][threadIdx.x];
    }
    __syncthreads();
  }
  if (row < numCRows && col < numCColumns) C[row * numCColumns + col] = Cvalue;
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA =
      ( float * )wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
  hostB =
      ( float * )wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC =
      ( float * )malloc(numCRows * numCColumns * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int size_a = numARows * numAColumns * sizeof(float);
  int size_b = numBRows * numBColumns * sizeof(float);
  int size_c = numCRows * numCColumns * sizeof(float);
  hipMalloc((void **) &deviceA, size_a);
  hipMalloc((void **) &deviceB, size_b);
  hipMalloc((void **) &deviceC, size_c);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, size_a, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, size_b, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid((numCColumns - 1)/TILE_WIDTH + 1, (numCRows - 1)/TILE_WIDTH + 1, 1);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC,
                                              numARows, numAColumns,
                                              numBRows, numBColumns,
                                              numCRows, numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, size_c, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
